

extern "C" 
void cudaInit(int rank, const double* localArray, size_t localArraySize, double** dLocalArray, double** dResult){
    hipError_t cE;
    if((cE = hipGetDeviceCount(&cudaDeviceCount)) != hipSuccess){
        printf("Unable to determine cuda device count, error is %d, count is %d\n", cE, cudaDeviceCount);
        exit(-1);
    }
    if((cE = hipSetDevice(rank % cudaDeviceCount)) != hipSuccess){
        printf("Unable to have rank %d set to cuda device %d, error is %d \n", 
                rank, (rank%cudaDeviceCount), cE);
        exit(-1);
    }
    hipMallocManaged((void**)&deviceLocalArray, localArraySize*sizeof(double));
    hipMemcpy(dbin2, localArray, bits*sizeof(int), hipMemcpyHostToDevice);
}

extern "C"
void cudaReduce(const double* dLocalArray, double* result, uint64_t* startTime, uint64_t* endTime){
    
}